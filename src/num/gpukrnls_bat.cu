#include "hip/hip_runtime.h"
/* Copyright 2023. TU Graz. Institute of Biomedical Imaging.
 * All rights reserved. Use of this source code is governed by
 * a BSD-style license which can be found in the LICENSE file.
 *
 * Authors: Moritz Blumenthal
 */

#include <stdio.h>
#include <stdbool.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include "misc/debug.h"
#include "misc/misc.h"

#include "num/gpuops.h"
#include "num/gpukrnls.h"
#include "num/multind.h"


static dim3 getBlockSize2(long Bi, long Bo, const void* func)
{
	int block[3] = { 1, 1, 1};

	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func));
	int threads = attr.maxThreadsPerBlock;

	block[0] = 1;
	block[1] = 1;

	while ((threads >= 2) && (block[0] < Bi)) {

		block[0] *= 2;
		threads /= 2;
	}

	while ((threads >= 2) && (block[1] < Bo)) {

		block[1] *= 2;
		threads /= 2;
	}

	return dim3(block[0], block[1], block[2]);
}

static long gridsize_int(long N, int blocksize)
{
	return MIN(65535, (N + blocksize - 1) / blocksize); // 65535 is maximum for y and z dim
}

static dim3 getGridSize2(long Bi, long Bo, const void* func)
{
	int block[3] = { 1, 1, 1};

	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func));
	int threads = attr.maxThreadsPerBlock;

	block[0] = 1;
	block[1] = 1;

	while ((threads >= 2) && (block[0] < Bi)) {

		block[0] *= 2;
		threads /= 2;
	}

	while ((threads >= 2) && (block[1] < Bo)) {

		block[1] *= 2;
		threads /= 2;
	}

	return dim3(gridsize_int(Bi, block[0]), gridsize_int(Bo, block[1]), 1);
}



__global__ static void kern_xpay_bat(long Bi, long N, long Bo, const float* _beta, hipFloatComplex* _a, const hipFloatComplex* _x)
{
	long bi_sta = threadIdx.x + blockDim.x * blockIdx.x;
	long bi_str = blockDim.x * gridDim.x;

	long bo_sta = threadIdx.y + blockDim.y * blockIdx.y;
	long bo_str = blockDim.y * gridDim.y;

	for (long bi = bi_sta; bi < Bi; bi += bi_str) {
		for (long bo = bo_sta; bo < Bo; bo += bo_str) {

			float beta = _beta[bi + Bi * bo];

			for (long i = 0; i < N; i++) {

				long idx = bi + Bi * i + Bi * N * bo;

				hipFloatComplex x = _x[idx];
				hipFloatComplex a = _a[idx];

				a.x = a.x * beta + x.x;
				a.y = a.y * beta + x.y;

				_a[idx] = a;
			}
		}
	}
}

extern "C" void cuda_xpay_bat(long Bi, long N, long Bo, const float* beta, float* a, const float* x)
{
	dim3 blockDim = getBlockSize2(Bi, Bo, (const void*)kern_xpay_bat);
	dim3 gridDim = getGridSize2(Bi, Bo, (const void*)kern_xpay_bat);

	kern_xpay_bat<<<gridDim, blockDim>>>(Bi, N, Bo, beta, (hipFloatComplex*) a, (const hipFloatComplex*)x);
	CUDA_KERNEL_ERROR;
}

__global__ static void kern_axpy_bat(long Bi, long N, long Bo, hipFloatComplex* _a, const float* _alpha, const hipFloatComplex* _x)
{
	long bi_sta = threadIdx.x + blockDim.x * blockIdx.x;
	long bi_str = blockDim.x * gridDim.x;

	long bo_sta = threadIdx.y + blockDim.y * blockIdx.y;
	long bo_str = blockDim.y * gridDim.y;

	for (long bi = bi_sta; bi < Bi; bi += bi_str) {
		for (long bo = bo_sta; bo < Bo; bo += bo_str) {

			float alpha = _alpha[bi + Bi * bo];

			for (long i = 0; i < N; i++) {

				long idx = bi + Bi * i + Bi * N * bo;

				hipFloatComplex x = _x[idx];
				hipFloatComplex a = _a[idx];

				a.x = a.x + x.x * alpha;
				a.y = a.y + x.y * alpha;

				_a[idx] = a;
			}
		}
	}
}


extern "C" void cuda_axpy_bat(long Bi, long N, long Bo, float* a, const float* alpha, const float* x)
{
	dim3 blockDim = getBlockSize2(Bi, Bo, (const void*)kern_axpy_bat);
	dim3 gridDim = getGridSize2(Bi, Bo, (const void*)kern_axpy_bat);

	kern_axpy_bat<<<gridDim, blockDim>>>(Bi, N, Bo, (hipFloatComplex*) a, alpha, (const hipFloatComplex*)x);
	CUDA_KERNEL_ERROR;
}


__global__ static void kern_dot_bat(long Bi, long N, long Bo, float* dst, const hipFloatComplex* _src1, const hipFloatComplex* _src2)
{
	long bi_sta = threadIdx.x + blockDim.x * blockIdx.x;
	long bi_str = blockDim.x * gridDim.x;

	long bo_sta = threadIdx.y + blockDim.y * blockIdx.y;
	long bo_str = blockDim.y * gridDim.y;

	for (long bi = bi_sta; bi < Bi; bi += bi_str) {
		for (long bo = bo_sta; bo < Bo; bo += bo_str) {

			double ret = 0;

			for (long i = 0; i < N; i++) {

				long idx = bi + Bi * i + Bi * N * bo;

				hipFloatComplex src1 = _src1[idx];
				hipFloatComplex src2 = _src2[idx];

				ret += src1.x * src2.x;
				ret += src1.y * src2.y;
			}

			dst[bi + Bi * bo] = ret;
		}
	}
}

extern "C" void cuda_dot_bat(long Bi, long N, long Bo, float* dst, const float* x, const float* y)
{
	dim3 blockDim = getBlockSize2(Bi, Bo, (const void*)kern_dot_bat);
	dim3 gridDim = getGridSize2(Bi, Bo, (const void*)kern_dot_bat);

	kern_dot_bat<<<gridDim, blockDim>>>(Bi, N, Bo, dst, (const hipFloatComplex*)x, (const hipFloatComplex*)y);
	CUDA_KERNEL_ERROR;
}
